#include <hip/hip_runtime.h>

#include <algorithm>
#include <array>
#include <iostream>
#include <memory>
#include <numeric>

#include "common/helper_cuda.hpp"
#include "init.cuh"
#include "one_sweep.cuh"

constexpr auto kRadix = 256;  // fixed for 32-bit unsigned int
constexpr auto kRadixPasses = 4;

[[nodiscard]] constexpr int GlobalHistThreadBlocks(const int size) {
  return 1;  // was 2048
  // return 2048;
}

[[nodiscard]] constexpr int BinningThreadBlocks(const int size) {
  // looks like we want to process 15 items per thread
  // and since 512 threads was used, we have
  constexpr auto partition_size = 7680;
  return size / partition_size;
}

constexpr auto kLaneCount = 32;       // fixed for NVIDIA GPUs
constexpr auto kGlobalHistWarps = 8;  // configurable
constexpr auto kDigitBinWarps = 16;   // configurable

// 8x32=256 threads
const dim3 kGlobalHistDim(kLaneCount, kGlobalHistWarps, 1);

// 16x32=512 threads
const dim3 kDigitBinDim(kLaneCount, kDigitBinWarps, 1);

struct RadixSortData {
  explicit RadixSortData(const int n) : n(n) {
    checkCudaErrors(hipMallocManaged(&u_sort, n * sizeof(unsigned int)));
    checkCudaErrors(hipMallocManaged(&u_sort_alt, n * sizeof(unsigned int)));
    checkCudaErrors(
        hipMallocManaged(&u_index, kRadixPasses * sizeof(unsigned int)));
    checkCudaErrors(hipMallocManaged(
        &u_global_histogram, kRadix * kRadixPasses * sizeof(unsigned int)));
    for (auto& pass_histogram : u_pass_histogram) {
      checkCudaErrors(hipMallocManaged(
          &pass_histogram,
          kRadix * BinningThreadBlocks(n) * sizeof(unsigned int)));
    }
  }

  ~RadixSortData() {
    checkCudaErrors(hipFree(u_sort));
    checkCudaErrors(hipFree(u_sort_alt));
    checkCudaErrors(hipFree(u_index));
    checkCudaErrors(hipFree(u_global_histogram));
    for (const auto& pass_histogram : u_pass_histogram) {
      checkCudaErrors(hipFree(pass_histogram));
    }
  }

  [[nodiscard]] bool IsSorted(const int n) const {
    return std::is_sorted(u_sort, u_sort + n);
  }

  void InitRandom(const int seed) const {
    constexpr auto block_size = 768;
    const auto num_blocks = (n + block_size - 1) / block_size;
    k_InitRandom<<<num_blocks, block_size>>>(u_sort, n, seed);
    checkCudaErrors(hipDeviceSynchronize());
  }

  void DispatchGlobalHistogram() const {
    k_GlobalHistogram<<<GlobalHistThreadBlocks(n), kGlobalHistDim>>>(
        u_sort, u_global_histogram, n);
  }

  void DispatchDigitBinning(const int pass) const {
    unsigned int* input;
    unsigned int* output;

    if (pass % 2 == 0) {
      input = u_sort;
      output = u_sort_alt;
    } else {
      input = u_sort_alt;
      output = u_sort;
    }

    k_DigitBinning<<<BinningThreadBlocks(n), kDigitBinDim>>>(
        u_global_histogram,
        input,
        output,
        u_pass_histogram[pass],
        u_index,
        n,
        pass * 8);
  }

  int n;
  unsigned int* u_sort;
  unsigned int* u_sort_alt;
  unsigned int* u_index;
  unsigned int* u_global_histogram;
  std::array<unsigned int*, kRadixPasses> u_pass_histogram;
};

int main(const int argc, const char* argv[]) {
  constexpr int size_exponent = 28;
  int n = 1 << size_exponent;  // 256M elements

  if (argc > 1) {
    n = std::strtol(argv[1], nullptr, 10);
  }

  // check n > 8096, and n is smaller than 2^28
  if (n < 8096 || n > (1 << 28)) {
    std::cerr << "n must be between 8096 and 2^28\n";
    return 1;
  }

  std::cout << "n = " << n << '\n';

  const auto data_ptr = std::make_unique<RadixSortData>(n);

  std::cout << "initializing...\n";
  constexpr auto seed = 114514;
  data_ptr->InitRandom(seed);

  auto result = data_ptr->IsSorted(n);
  std::cout << "Before sorting: Is sorted ? " << std::boolalpha << result
            << '\n';

  std::cout << "start sorting...\n";

  data_ptr->DispatchGlobalHistogram();
  data_ptr->DispatchDigitBinning(0);
  data_ptr->DispatchDigitBinning(1);
  data_ptr->DispatchDigitBinning(2);
  data_ptr->DispatchDigitBinning(3);

  checkCudaErrors(hipDeviceSynchronize());

  result = data_ptr->IsSorted(n);
  std::cout << "After sorting: Is sorted ? " << std::boolalpha << result
            << '\n';

  return 0;
}
